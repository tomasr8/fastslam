#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#ifndef M_PI
#define M_PI 3.14159265359
#endif

#define PARTICLE_SIZE <<PARTICLE_SIZE>>

__device__ float* get_particle(float *particles, int i) {
    return (particles + PARTICLE_SIZE*i);
}

// Manual extern "C" to stop name mangling shenanigans
// Otherwise doesn't compile because hiprand complains
extern "C" {

// Based on https://stackoverflow.com/questions/46169633/how-to-generate-random-number-inside-pycuda-kernel    
// Each thread has a random state
__device__ hiprandState_t* states[<<N_PARTICLES>>];


// This function is only called once to initialize the rngs.
__global__ void init_rng(int seed)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t* s = new hiprandState_t;
    hiprand_init(seed, i, 0, s);
    states[i] = s;
}


__global__ void predict_from_imu(float *particles,
    float x, float y, float theta, float sigma_x, float sigma_y, float sigma_theta) {
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    float *particle = get_particle(particles, i);
    // hiprand_normal() samples from standard normal
    // to get a general N(mu, sigma), we use Y = mu + sigma*X,
    // though in our case mu=0.
    particle[0] = x + sigma_x * hiprand_normal(states[i]);
    particle[1] = y + sigma_y * hiprand_normal(states[i]);
    particle[2] = theta + sigma_theta * hiprand_normal(states[i]);
}

// Moves particles based on the control input and movement model.
__global__ void predict_from_model(float *particles, float ua, float ub, float sigma_a, float sigma_b, float dt) {
    if(ua == 0.0 && ub == 0.0) {
        return;
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    float *particle = get_particle(particles, i);
    // hiprand_normal() samples from standard normal
    // to get a general N(mu, sigma), we use Y = mu + sigma*X,
    // though in our case mu=0.
    particle[2] += (ua * dt) + sigma_a * hiprand_normal(states[i]);
    particle[2] = fmod(particle[2], (float)(2*M_PI));

    float dist = (ub * dt) + sigma_b * hiprand_normal(states[i]);
    particle[0] += cos(particle[2]) * dist;
    particle[1] += sin(particle[2]) * dist;
}
}