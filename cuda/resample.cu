#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define MIN(a,b) (((a)<(b))?(a):(b))

#define PARTICLE_SIZE <<PARTICLE_SIZE>>

__device__ float* get_particle(float *particles, int i) {
    return (particles + PARTICLE_SIZE*i);
}

/*
 * Copies particles from one memory block to another based on indices
 * given by systematic resampling.
 *
 * The systematic resampling is handled on the host. I haven't yet figured out
 * how to do it efficiently on the device and the CPU implementation is fast enough.
 */
__global__ void resample(
    float *old_particles, float *new_particles, int *idx, int block_size, int n_particles)
{

    // *idx is a mapping where i is the index of the new particle and
    // idx[i] is the index of the old particle.

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    // int particle_size = 6 + 7*((int)old_particles[4]);
    int id_min = thread_id*block_size;
    int id_max = thread_id*block_size + (block_size - 1);
    // int id_max = MIN(thread_id*block_size + (block_size - 1), n_particles - 1);


    for(int i = id_min; i <= id_max; i++) {
        float *old_particle = get_particle(old_particles, idx[i]);
        float *new_particle = get_particle(new_particles, i);

        for(int k = 0; k < PARTICLE_SIZE; k++) {
            new_particle[k] = old_particle[k];
        }

        new_particle[3] = 1.0/n_particles;
    }
}

// __global__ void resample(
//     float *particles, float *new_particles, int block_size, int n_particles, float random)
// {
//     // return;
//     int block_id = blockIdx.x+ blockIdx.y * gridDim.x;
//     int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

//     // if(thread_id == 0) {
//     //     printf("dfgdfgd\n");
//     // }

//     int particle_size = 6 + 7*((int)particles[4]);
//     int id_min = thread_id*block_size;
//     int id_max = MIN(thread_id*block_size + (block_size - 1), n_particles - 1);


//     int size = (n_particles) * sizeof(float);
//     float *cumsum;

//     cumsum = (float *)malloc(size);
//     cumsum[0] = get_particle(particles, 0)[3];

//     for(int i = 1; i < n_particles; i++) {
//         cumsum[i] = cumsum[i-1] + get_particle(particles, i)[3];
//     }

//     cumsum[n_particles-1] = 1.0;

//     int i = 0;
//     int j = 0;
//     while(i < n_particles) {
//         if( ((i + random)/n_particles) < cumsum[j] ) {

//             if(i >= id_min && i <= id_max) {
//                 float *new_particle = get_particle(new_particles, i);
//                 float *old_particle = get_particle(particles, j);

//                 // memcpy(new_particle, old_particle, particle_size);

//             }

//             if(i > id_max) {
//                 break;
//             }

//             i += 1;
//         } else {
//             j += 1;
//         }
//     }

//     free(cumsum);
// }