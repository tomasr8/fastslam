#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define PARTICLE_SIZE <<PARTICLE_SIZE>>
#define N_PARTICLES <<N_PARTICLES>>
#define THREADS <<THREADS>>

__device__ float* get_particle(float *particles, int i) {
    return (particles + PARTICLE_SIZE*i);
}

/*
 * Sums particle weights.
 * Needs to run in a single block.
 */
 __global__ void sum_weights(float *particles, double *out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0;

    for (int i = idx; i < N_PARTICLES; i += THREADS) {
        float *particle = get_particle(particles, i);
        sum += (double)particle[3];
    }

    __shared__ double r[THREADS];
    r[idx] = sum;
    __syncthreads();

    for (int size = THREADS/2; size > 0; size /= 2) {
        if (idx < size) {
            r[idx] += r[idx + size];
        }
        __syncthreads();
    }

    if (idx == 0) {
        *out = r[0];
    }
}

/*
 * Rescales particle weights so that \sum_i w_i = 1
 */
 __global__ void divide_weights(float *particles, double *s) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = (float)s[0];
    float *particle = get_particle(particles, i);
    particle[3] /= sum;
}