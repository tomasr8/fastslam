#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define PARTICLE_SIZE <<PARTICLE_SIZE>>
#define N_PARTICLES <<N_PARTICLES>>
#define THREADS <<THREADS>>

__device__ float* get_particle(float *particles, int i) {
    return (particles + PARTICLE_SIZE*i);
}

/*
 * Sums particle weights.
 * Needs to run in a single block.
 */
 __global__ void sum_weights(float *particles, float *out) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int idx = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    float sum = 0;

    for (int i = idx; i < N_PARTICLES; i += THREADS) {
        float *particle = get_particle(particles, i);
        sum += particle[3];
    }

    __shared__ float r[THREADS];
    r[idx] = sum;
    __syncthreads();

    for (int size = THREADS/2; size>0; size/=2) { //uniform
        if (idx<size) {
            r[idx] += r[idx+size];
        }
        __syncthreads();
    }

    if (idx == 0) {
        *out = r[0];
    }
}

/*
 * Rescales particle weights so that \sum_i w_i = 1
 */
 __global__ void divide_weights(float *particles, float *s) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    float sum = s[0];
    float *particle = get_particle(particles, thread_id);
    particle[3] /= sum;
}