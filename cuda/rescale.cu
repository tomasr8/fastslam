#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define PARTICLE_SIZE <<PARTICLE_SIZE>>

__device__ float* get_particle(float *particles, int i) {
    return (particles + PARTICLE_SIZE*i);
}

__global__ void rescale(float *particles, int n_particles) {
    float s = 0;

    for(int i = 0; i < n_particles; i++) {
        float *particle = get_particle(particles, i);
        s += particle[3];
    }

    s += 1.e-30;

    for(int i = 0; i < n_particles; i++) {
        float *particle = get_particle(particles, i);
        particle[3] /= s;
    }
}