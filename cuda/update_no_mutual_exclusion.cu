
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define M_PI 3.14159265359
#define MIN(a,b) (((a)<(b))?(a):(b))


typedef struct 
{
    float (*measurements)[2];
    int n_measurements;
    float *measurement_cov;
} landmark_measurements;

__device__ float vecnorm(float *v) {
    return sqrt(v[0]*v[0] + v[1]*v[1]);
}

__device__ bool in_sensor_range(float *position, float *landmark, float range, float fov) {
    float x = position[0];
    float y = position[1];
    float theta = position[2];
    float lx = landmark[0];
    float ly = landmark[1];

    float va[] = {lx - x, ly - y};
    float vb[] = {range * cos(theta), range * sin(theta)};

    if(vecnorm(va) > range) {
        return false;
    }

    float angle = acos(
        (va[0]*vb[0] + va[1]*vb[1])/(vecnorm(va)*vecnorm(vb))
    );

    if(angle <= (fov/2)) {
        return true;
    } else {
        return false;
    }
}

__device__ float* get_particle(float *particles, int i) {
    int max_landmarks = (int)particles[4];
    return (particles + (6 + 7*max_landmarks)*i);
}

__device__ float* get_mean(float *particle, int i)
{
    return (particle + 6 + 2*i);
}

__device__ float* get_cov(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    return (particle + 6 + 2*max_landmarks + 4*i);
}

__device__ float* get_landmark_prob(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    return (particle + 6 + 6*max_landmarks + i);
}

__device__ void increment_landmark_prob(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    float *prob = (particle + 6 + 6*max_landmarks + i);
    prob[0] += 1.0;
}

__device__ void decrement_landmark_prob(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    float *prob = (particle + 6 + 6*max_landmarks + i);
    prob[0] -= 1.0;
}

__device__ int get_n_landmarks(float *particle)
{
    return (int)particle[5];
}

__device__ void add_landmark(float *particle, float mean[2], float *cov)
{
    int n_landmarks = (int)particle[5];
    particle[5] = (float)(n_landmarks + 1);

    float *new_mean = get_mean(particle, n_landmarks);
    float *new_cov = get_cov(particle, n_landmarks);
    float *new_prob = get_landmark_prob(particle, n_landmarks);

    new_mean[0] = mean[0];
    new_mean[1] = mean[1];

    new_cov[0] = cov[0];
    new_cov[1] = cov[1];
    new_cov[2] = cov[2];
    new_cov[3] = cov[3];

    new_prob[0] = 1.0;
}

__device__ void remove_landmark(float *particle, int i)
{
    int n_landmarks = (int)particle[5];

    for(int j = i + 1; j < n_landmarks; j++) {
        float *prob_a = get_landmark_prob(particle, j - 1);
        float *prob_b = prob_a + 1;

        prob_a[0] = prob_b[0];
    }
    
    for(int j = i + 1; j < n_landmarks; j++) {
        float *cov_a = get_cov(particle, j - 1);
        float *cov_b = cov_a + 4;

        cov_a[0] = cov_b[0];
        cov_a[1] = cov_b[1];
        cov_a[2] = cov_b[2];
        cov_a[3] = cov_b[3];
    }

    for(int j = i + 1; j < n_landmarks; j++) {
        float *mean_a = get_mean(particle, j - 1);
        float *mean_b = mean_a + 2;

        mean_a[0] = mean_b[0];
        mean_a[1] = mean_b[1];
    }

    particle[5] = (float)(n_landmarks - 1);
}

__device__ void vecmul(float *A, float *u, float *v)
{
    float a = A[0];
    float b = A[1];
    float c = A[2];
    float d = A[3];

    float e = u[0];
    float f = v[1];

    v[0] = a*e + b*f;
    v[1] = c*e + d*f;
}

__device__ void matmul(float *A, float *B, float *C)
{
    float a = A[0];
    float b = A[1];
    float c = A[2];
    float d = A[3];

    float e = B[0];
    float f = B[1];
    float g = B[2];
    float h = B[3];

    C[0] = a*e + b*g;
    C[1] = a*f + b*h;
    C[2] = c*e + d*g;
    C[3] = c*f + d*h;
}

__device__ void pinv(float *A, float *B)
{
    float a = A[0];
    float b = A[1];
    float c = A[2];
    float d = A[3];

    float e = a*a + c*c;
    float f = a*b + c*d;
    float g = a*b + c*d;
    float h = b*b + d*d;

    float scalar = 1/(e*h - f*g);
    float e_i = scalar * h;
    float f_i = scalar * (-f);
    float g_i = scalar * (-g);
    float h_i = scalar * e;

    B[0] = e_i*a + f_i*b;
    B[1] = e_i*c + f_i*d;
    B[2] = g_i*a + h_i*b;
    B[3] = g_i*c + h_i*d;
}

__device__ float pdf(float *x, float *mean, float* cov)
{
    float a = cov[0];
    float b = cov[1];

    float logdet = log(a*a - b*b);

    float root = sqrt(2.0)/2.0;
    float e = root * (1.0/sqrt(a-b));
    float f = root * (1.0/sqrt(a+b));

    float m = x[0] - mean[0];
    float n = x[1] - mean[1];

    float maha = 2*(m*m*e*e + n*n*f*f);
    float log2pi = log(2 * M_PI);
    return exp(-0.5 * (2*log2pi + maha + logdet));
}

__device__ void add_measurements_as_landmarks(float *particle, landmark_measurements *measurements)
{
    int n_measurements = measurements->n_measurements;
    float *measurement_cov = measurements->measurement_cov;

    for(int i = 0; i < n_measurements; i++) {
        float x = particle[0];
        float y = particle[1];
        float measurement[] = {
            x + measurements->measurements[i][0],
            y + measurements->measurements[i][1]
        };

        add_landmark(particle, measurement, measurement_cov);
    }
}

__device__ void add_measurement_as_landmark(float *particle, float *measurement, float *measurement_cov)
{
    float x = particle[0];
    float y = particle[1];
    float landmark[] = {
        x + measurement[0],
        y + measurement[1]
    };

    add_landmark(particle, landmark, measurement_cov);
}

__device__ float compute_dist(float *particle, int i, float *measurement, float *measurement_cov)
{
    float pos[] = { particle[0], particle[1] };
    float *landmark_cov = get_cov(particle, i);
    float *landmark = get_mean(particle, i);

    float measurement_predicted[] = {
        landmark[0] - pos[0], landmark[1] - pos[1]
    };

    float cov[4] = {
        landmark_cov[0] + measurement_cov[0],
        landmark_cov[1] + measurement_cov[1],
        landmark_cov[2] + measurement_cov[2],
        landmark_cov[3] + measurement_cov[3]
    };

    return pdf(measurement_predicted, measurement, cov);
}


__device__ void update_landmarks(int id, float *particle, landmark_measurements *measurements, int *in_range, int *n_matches, float range, float fov, float thresh)
{
    float *measurement_cov = measurements->measurement_cov;
    int n_measurements = measurements->n_measurements;

    // int *in_range = (int*)malloc(250 * sizeof(int));
    // int *n_matches = (int*)malloc(250 * sizeof(int));

    float x = particle[0];
    float y = particle[1];
    int n_landmarks = get_n_landmarks(particle);

    int n_in_range = 0;
    for(int i = 0; i < n_landmarks; i++) {
        n_matches[i] = 0;


        // in_range[n_in_range] = i;
        // n_in_range++;

        float *mean = get_mean(particle, i);
        if(in_sensor_range(particle, mean, range + 1, fov + 0.2)) {
            in_range[n_in_range] = i;
            n_in_range++;
        }
    }

    // if(id == 0) {
    //     printf("in range: %d/%d \n", n_in_range, n_landmarks);
    // }

    for(int i = 0; i < n_measurements; i++) {
        float best = -1;
        int best_idx = -1;

        for(int j = 0; j < n_in_range; j++) {
            float dist = compute_dist(particle, in_range[j], measurements->measurements[i], measurement_cov);

            if(dist > thresh && dist > best) {
                best = dist;
                best_idx = in_range[j];
            }
        }

        if(best_idx != -1) {
            n_matches[best_idx]++;
        }


        if(best_idx != -1) {
            float *mean = get_mean(particle, best_idx);
            float mean_x = mean[0];
            float mean_y = mean[1];

            float measurement_predicted[2] = { mean_x - x, mean_y - y };
            float residual[2] = {
                measurements->measurements[i][0] - measurement_predicted[0],
                measurements->measurements[i][1] - measurement_predicted[1]
            };

            float *cov = get_cov(particle, best_idx);

            float Q[4] = {
                cov[0] + measurement_cov[0],
                cov[1] + measurement_cov[1],
                cov[2] + measurement_cov[2],
                cov[3] + measurement_cov[3]
            };

            float K[4] = { 0, 0, 0, 0 };
            float Q_inv[4] = { 0, 0, 0, 0 };
            pinv(Q, Q_inv);
            matmul(cov, Q_inv, K);

            float K_residual[] = { 0, 0 };
            vecmul(K, residual, K_residual);
            mean[0] += K_residual[0];
            mean[1] += K_residual[1];

            float new_cov[] = { 1 - K[0], K[1], K[2], 1 - K[3] };
            matmul(new_cov, cov, new_cov);
            cov[0] = new_cov[0];
            cov[1] = new_cov[1];
            cov[2] = new_cov[2];
            cov[3] = new_cov[3];

            particle[3] *= pdf(measurements->measurements[i], measurement_predicted, Q);
            increment_landmark_prob(particle, best_idx);
        } else {
            add_measurement_as_landmark(particle, measurements->measurements[i], measurement_cov);
        }
    }

    // for(int i = n_in_range - 1; i > 0; i--) {
    //     int idx = in_range[i];
    //     if(n_matches[idx] == 0) {
    //         decrement_landmark_prob(particle, idx);
    //         float prob = get_landmark_prob(particle, idx)[0];
    //         if(prob < 0) {
    //             remove_landmark(particle, idx);
    //         }
    //     } 
    // }
}

__global__ void update(
    float *particles, int block_size, int *scratchpad_mem, int scratchpad_size, float measurements_array[][2], int n_particles, int n_measurements,
    float *measurement_cov, float threshold, float range, float fov)
{
    // int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(n_measurements == 0) {
        return;
    }

    int block_id = blockIdx.x+ blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    // int *in_range = (int*)malloc(250 * sizeof(int));
    // int *n_matches = (int*)malloc(250 * sizeof(int));

    int *scratchpad = scratchpad_mem + (thread_id * 500);
    int *in_range = scratchpad;
    int *n_matches = in_range + (250);

    landmark_measurements measurements;
    measurements.n_measurements = n_measurements;
    measurements.measurement_cov = measurement_cov;
    measurements.measurements = measurements_array;

    for(int k = 0; k < block_size; k++) {
        int particle_id = thread_id*block_size + k;
        if(particle_id >= n_particles) {
            return;
        }
        
        float *particle = get_particle(particles, particle_id);
        int n_landmarks = get_n_landmarks(particle);
    
        if(n_landmarks == 0) {
            add_measurements_as_landmarks(particle, &measurements);
            continue;
        }

        update_landmarks(particle_id, particle, &measurements, in_range, n_matches, range, fov, threshold);
    }
}