#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define M_PI 3.14159265359
#define MIN(a,b) (((a)<(b))?(a):(b))


typedef struct 
{
    float (*measurements)[2];
    int n_measurements;
    float *measurement_cov;
} landmark_measurements;

__device__ float vecnorm(float *v) {
    return sqrt(v[0]*v[0] + v[1]*v[1]);
}

__device__ bool in_sensor_range(float *position, float *landmark, float range, float fov) {
    float x = position[0];
    float y = position[1];
    float theta = position[2];
    float lx = landmark[0];
    float ly = landmark[1];

    float va[] = {lx - x, ly - y};
    float vb[] = {range * cos(theta), range * sin(theta)};

    if(vecnorm(va) > range) {
        return false;
    }

    float angle = acos(
        (va[0]*vb[0] + va[1]*vb[1])/(vecnorm(va)*vecnorm(vb))
    );

    if(angle <= (fov/2)) {
        return true;
    } else {
        return false;
    }
}

__device__ bool in_large_sensor_range(float *position, float *landmark, float range) {
    float x = position[0];
    float y = position[1];
    float lx = landmark[0];
    float ly = landmark[1];

    float dist_sq = (lx-x)*(lx-x) + (ly-y)*(ly-y);

    return dist_sq < range*range;
}

__device__ void to_coords(float *particle, float *in, float *out) {
    float x = particle[0];
    float y = particle[1];
    float theta = particle[2];

    float range = in[0];
    float bearing = in[1];

    out[0] = x + range * cos(bearing + theta);
    out[1] = y + range * sin(bearing + theta);
}

__device__ float* get_particle(float *particles, int i) {
    int max_landmarks = (int)particles[4];
    return (particles + (6 + 7*max_landmarks)*i);
}

__device__ float* get_mean(float *particle, int i)
{
    return (particle + 6 + 2*i);
}

__device__ float* get_cov(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    return (particle + 6 + 2*max_landmarks + 4*i);
}

__device__ float* get_landmark_prob(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    return (particle + 6 + 6*max_landmarks + i);
}

__device__ void increment_landmark_prob(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    float *prob = (particle + 6 + 6*max_landmarks + i);
    prob[0] += 1.0;
}

__device__ void decrement_landmark_prob(float *particle, int i)
{
    int max_landmarks = (int)particle[4];
    float *prob = (particle + 6 + 6*max_landmarks + i);
    prob[0] -= 1.0;
}

__device__ int get_n_landmarks(float *particle)
{
    return (int)particle[5];
}

__device__ void add_landmark(float *particle, float mean[2], float *cov)
{
    int n_landmarks = (int)particle[5];
    particle[5] = (float)(n_landmarks + 1);

    float *new_mean = get_mean(particle, n_landmarks);
    float *new_cov = get_cov(particle, n_landmarks);
    float *new_prob = get_landmark_prob(particle, n_landmarks);

    new_mean[0] = mean[0];
    new_mean[1] = mean[1];

    new_cov[0] = cov[0];
    new_cov[1] = cov[1];
    new_cov[2] = cov[2];
    new_cov[3] = cov[3];

    new_prob[0] = 1.0;
}

__device__ void remove_landmark(float *particle, int i)
{
    int n_landmarks = (int)particle[5];

    for(int j = i + 1; j < n_landmarks; j++) {
        float *prob_a = get_landmark_prob(particle, j - 1);
        float *prob_b = prob_a + 1;

        prob_a[0] = prob_b[0];
    }
    
    for(int j = i + 1; j < n_landmarks; j++) {
        float *cov_a = get_cov(particle, j - 1);
        float *cov_b = cov_a + 4;

        cov_a[0] = cov_b[0];
        cov_a[1] = cov_b[1];
        cov_a[2] = cov_b[2];
        cov_a[3] = cov_b[3];
    }

    for(int j = i + 1; j < n_landmarks; j++) {
        float *mean_a = get_mean(particle, j - 1);
        float *mean_b = mean_a + 2;

        mean_a[0] = mean_b[0];
        mean_a[1] = mean_b[1];
    }

    particle[5] = (float)(n_landmarks - 1);
}

__device__ void vecmul(float *A, float *u, float *v)
{
    float a = A[0];
    float b = A[1];
    float c = A[2];
    float d = A[3];

    float e = u[0];
    float f = v[1];

    v[0] = a*e + b*f;
    v[1] = c*e + d*f;
}

__device__ void matmul(float *A, float *B, float *C)
{
    float a = A[0];
    float b = A[1];
    float c = A[2];
    float d = A[3];

    float e = B[0];
    float f = B[1];
    float g = B[2];
    float h = B[3];

    C[0] = a*e + b*g;
    C[1] = a*f + b*h;
    C[2] = c*e + d*g;
    C[3] = c*f + d*h;
}

__device__ void matmul_jacobian(float *H, float *E, float *R, float *S)
{
    float a = H[0];
    float b = H[1];
    float c = H[2];
    float d = H[3];

    float Ht[] = {
        a, c,
        b, d
    };

    matmul(Ht, E, S);
    matmul(S, H, S);

    S[0] += R[0];
    S[1] += R[1];
    S[2] += R[2];
    S[3] += R[3];
}

__device__ void pinv(float *A, float *B)
{
    float a = A[0];
    float b = A[1];
    float c = A[2];
    float d = A[3];

    float scalar = 1/(a*d - b*c);

    B[0] = scalar * d;
    B[1] = scalar * (-b);
    B[2] = scalar * (-c);
    B[3] = scalar * a;
}

__device__ float pdf(float *x, float *mean, float* cov)
{
    float a = cov[0];
    float b = cov[1];

    float logdet = log(a*a - b*b);

    float root = sqrt(2.0)/2.0;
    float e = root * (1.0/sqrt(a-b));
    float f = root * (1.0/sqrt(a+b));

    float m = x[0] - mean[0];
    float n = x[1] - mean[1];

    float maha = 2*(m*m*e*e + n*n*f*f);
    float log2pi = log(2 * M_PI);
    return exp(-0.5 * (2*log2pi + maha + logdet));
}

__device__ float mahalanobis(float *u, float *v, float* cov) {
    float inv[4] = { 0, 0, 0, 0 };
    pinv(cov, inv);

    float x = u[0] - v[0];
    float y = u[1] - v[1];

    return x*x*cov[0] + x*y*(cov[1] + cov[2]) + y*y*cov[3];
}

__device__ void add_measurements_as_landmarks(float *particle, landmark_measurements *measurements)
{
    int n_measurements = measurements->n_measurements;
    float *measurement_cov = measurements->measurement_cov;

    for(int i = 0; i < n_measurements; i++) {
        add_measurement_as_landmark(particle, measurements->measurements[i], measurement_cov);
    }
}

__device__ void add_measurement_as_landmark(float *particle, float *measurement, float *measurement_cov)
{
    float pos[] = { particle[0], particle[1] };
    float theta = particle[2];
    float landmark[] = {0, 0};
    to_coords(particle, measurement, landmark);

    float q = (landmark[0] - pos[0])*(landmark[0] - pos[0]) + (landmark[1] - pos[1])*(landmark[1] - pos[1]);

    float H[] = {
        (landmark[0] - pos[0])/sqrt(q), (landmark[1] - pos[1])/sqrt(q),
        (landmark[0] - pos[0])/q, -(landmark[1] - pos[1])/q
    };

    float Ht[] = {
        H[0], H[2],
        H[1], H[3]
    };

    pinv(H, H);

    float S[] = {
        0, 0, 0, 0
    };

    matmul(H, measurement_cov, S);
    matmul(S, Ht, S);

    add_landmark(particle, landmark, S);
}

__device__ float compute_dist(float *particle, int i, float *measurement, float *measurement_cov)
{
    float pos[] = { particle[0], particle[1] };
    float theta = particle[2];
    float *landmark_cov = get_cov(particle, i);
    float *landmark = get_mean(particle, i);

    float q = (landmark[0] - pos[0])*(landmark[0] - pos[0]) + (landmark[1] - pos[1])*(landmark[1] - pos[1]);

    float measurement_predicted[] = {
        sqrt(q), atan2(landmark[1] - pos[1], landmark[0] - pos[0]) - theta
    };

    float H[] = {
        (landmark[0] - pos[0])/sqrt(q), (landmark[1] - pos[1])/sqrt(q),
        -(landmark[1] - pos[1])/q, (landmark[0] - pos[0])/q,
    };

    float S[] = {
        0, 0, 0, 0
    };

    matmul_jacobian(H, landmark_cov, measurement_cov, S);

    return pdf(measurement_predicted, measurement, S);
    // return mahalanobis(measurement_predicted, measurement, cov);
}


__device__ void update_landmarks(int id, float *particle, landmark_measurements *measurements, int *in_range, int *n_matches, float range, float fov, float thresh)
{
    float *measurement_cov = measurements->measurement_cov;
    int n_measurements = measurements->n_measurements;

    float x = particle[0];
    float y = particle[1];
    int n_landmarks = get_n_landmarks(particle);

    int n_in_range = 0;
    for(int i = 0; i < n_landmarks; i++) {
        n_matches[i] = 0;
        float *mean = get_mean(particle, i);
        in_range[n_in_range] = i;
        n_in_range++;
        // if(in_large_sensor_range(particle, mean, range + 2)) {
        //     in_range[n_in_range] = i;
        //     n_in_range++;
        // }
    }

    for(int i = 0; i < n_measurements; i++) {
        float best = -1;
        int best_idx = -1;

        for(int j = 0; j < n_in_range; j++) {
            float dist = compute_dist(particle, in_range[j], measurements->measurements[i], measurement_cov);

            if(dist > thresh && dist > best) {
                best = dist;
                best_idx = in_range[j];
            }
        }

        // if(id == 0) {
        //     printf("best dist: %f, thresh: %f\n", best, thresh);
        // }

        if(best_idx != -1) {
            n_matches[best_idx]++;
        }


        if(best_idx != -1) {
            float *landmark = get_mean(particle, best_idx);
            float pos[] = { particle[0], particle[1] };
            // float mean_x = mean[0];
            // float mean_y = mean[1];

            float q = (landmark[0] - pos[0])*(landmark[0] - pos[0]) + (landmark[1] - pos[1])*(landmark[1] - pos[1]);
            float measurement_predicted[] = {
                sqrt(q), atan2(landmark[1] - pos[1], landmark[0] - pos[0]) - theta
            };

            float residual[2] = {
                measurements->measurements[i][0] - measurement_predicted[0],
                measurements->measurements[i][1] - measurement_predicted[1]
            };

            float H[] = {
                (landmark[0] - pos[0])/sqrt(q), (landmark[1] - pos[1])/sqrt(q),
                -(landmark[1] - pos[1])/q, (landmark[0] - pos[0])/q,
            };
        
            float S[] = {
                0, 0, 0, 0
            };
        
            matmul_jacobian(H, landmark_cov, measurement_cov, S);
            float S_inv[] = {0, 0, 0, 0};
            pinv(S, S_inv);

            float *cov = get_cov(particle, best_idx);

            float Q[] = {0, 0, 0, 0};
            float K[] = { 0, 0, 0, 0 };
            matmul(cov, H, Q);
            matmul(Q, S_inv, K);

            float K_residual[] = { 0, 0 };
            vecmul(K, residual, K_residual);
            landmark[0] += K_residual[0];
            landmark[1] += K_residual[1];

            float KH[] = { 0, 0, 0, 0};
            matmul(K, H, KH);
            float new_cov[] = { 1 - KH[0], KH[1], KH[2], 1 - KH[3] };
            matmul(new_cov, cov, new_cov);
            cov[0] = new_cov[0];
            cov[1] = new_cov[1];
            cov[2] = new_cov[2];
            cov[3] = new_cov[3];

            particle[3] *= pdf(measurements->measurements[i], measurement_predicted, S);
            increment_landmark_prob(particle, best_idx);
        } else {
            add_measurement_as_landmark(particle, measurements->measurements[i], measurement_cov);
        }
    }

    // for(int i = n_in_range - 1; i > 0; i--) {
    //     int idx = in_range[i];
    //     if(n_matches[idx] == 0) {
    //         decrement_landmark_prob(particle, idx);
    //         float prob = get_landmark_prob(particle, idx)[0];
    //         if(prob < 0) {
    //             remove_landmark(particle, idx);
    //         }
    //     } 
    // }
}

// __shared__ int scratchpad_mem[<<SCRATCHPAD_SIZE>>];

__global__ void update(
    float *particles, int block_size, int *scratchpad_mem, int scratchpad_size, float measurements_array[][2], int n_particles, int n_measurements,
    float *measurement_cov, float threshold, float range, float fov, int max_landmarks)
{
    // int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(n_measurements == 0) {
        return;
    }

    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    // if(thread_id == 0) {
    //     for(int i = 0; i < n_measurements; i++) {
    //         printf("%d [%f %f]\n", i, measurements_array[i][0], measurements_array[i][1]);
    //     }
    // }

    int *scratchpad = scratchpad_mem + (2 * thread_id * max_landmarks);
    int *in_range = scratchpad;
    int *n_matches = in_range + max_landmarks;

    landmark_measurements measurements;
    measurements.n_measurements = n_measurements;
    measurements.measurement_cov = measurement_cov;
    measurements.measurements = measurements_array;

    for(int k = 0; k < block_size; k++) {
        int particle_id = thread_id*block_size + k;
        if(particle_id >= n_particles) {
            return;
        }
        
        float *particle = get_particle(particles, particle_id);
        int n_landmarks = get_n_landmarks(particle);
    
        if(n_landmarks == 0) {
            add_measurements_as_landmarks(particle, &measurements);
            continue;
        }

        update_landmarks(particle_id, particle, &measurements, in_range, n_matches, range, fov, threshold);
    }
}