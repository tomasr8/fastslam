#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#define N_PARTICLES <<N_PARTICLES>>
#define PARTICLE_SIZE <<PARTICLE_SIZE>>
#define THREADS <<THREADS>>

__device__ float* get_particle(float *particles, int i) {
    return (particles + PARTICLE_SIZE*i);
}

// /*
//  * Extracts the particle weights and computes the estimated position.
//  * This code is not parallelized since it is reasonobly fast.
//  */
// __global__ void get_weights_and_mean_position(float *particles, int n_particles, float *weights, float *mean) {
//     float x = 0;
//     float y = 0;
//     float theta = 0;

//     for(int i = 0; i < n_particles; i++) {
//         float *particle = get_particle(particles, i);
//         weights[i] = particle[3];
//         x += particle[3] * particle[0];
//         y += particle[3] * particle[1];
//         theta += particle[3] * particle[2];
//     }

//     mean[0] = x;
//     mean[1] = y;
//     mean[2] = theta;
// }


/*
 * Extracts weights from particles.
 */
 __global__ void get_weights(float *particles, double *weights) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    float *particle = get_particle(particles, i);
    weights[i] = (double)particle[3];
}


/*
 * Calculates the mean position of all particles.
 * Needs to run in a single block.
 */
 __global__ void get_mean_position(float *particles, float *mean) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int idx = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    float x = 0;
    float y = 0;
    float theta = 0;

    for (int i = idx; i < N_PARTICLES; i += THREADS) {
        float *particle = get_particle(particles, i);
        x += particle[3] * particle[0];
        y += particle[3] * particle[1];
        theta += particle[3] * particle[2];
    }

    __shared__ float r_x[THREADS];
    __shared__ float r_y[THREADS];
    __shared__ float r_theta[THREADS];

    r_x[idx] = x;
    r_y[idx] = y;
    r_theta[idx] = theta;

    __syncthreads();

    for (int size = THREADS/2; size>0; size/=2) {
        if (idx<size) {
            r_x[idx] += r_x[idx+size];
            r_y[idx] += r_y[idx+size];
            r_theta[idx] += r_theta[idx+size];
        }
        __syncthreads();
    }

    if (idx == 0) {
        mean[0] = r_x[0];
        mean[1] = r_y[0];
        mean[2] = r_theta[0];
    }
}