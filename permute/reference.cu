
#include <hip/hip_runtime.h>
#include <stdbool.h>

__global__ void reset(int *d, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d[i] = n;
}

__global__ void prepermute(int *ancestors, int *d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    d[ancestors[i]] = i;

    // int *p = d + ancestors[i];
    // atomicMin(p, i);
}


// __global__ void permute_reference(int *ancestors, int *c, int *d, int block_size, int n) {
//     int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

//     for(int k = 0; k < block_size; k++) {
//         int i = thread_id*block_size + k;

//         int x = d[ancestors[i]];
//         if(x != i) {
//             x = i;
//             while(d[x] < n) {
//                 x = d[x];
//             }
//             d[x] = i;
//         }
    
//     }

//     __syncthreads();

//     for(int k = 0; k < block_size; k++) {
//         int i = thread_id*block_size + k;

//         c[i] = ancestors[d[i]];
//     }
// }


__global__ void permute_reference(int *ancestors, int *c, int *d, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int x = d[ancestors[i]];
    if(x != i) {
        x = i;
        while(d[x] < n) {
            x = d[x];
        }
        d[x] = i;
    }
}

__global__ void write_to_c(int *ancestors, int *c, int *d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    c[i] = ancestors[d[i]];
}